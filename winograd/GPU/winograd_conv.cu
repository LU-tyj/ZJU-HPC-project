#include "hip/hip_runtime.h"
#include "winograd.cuh"

__global__
void winograd_conv_kernel(const float* __restrict__ image,
                          const float* __restrict__ filter,
                          float* __restrict__ output,
                          int N, int C, int H, int W, int K, int outH, int outW)
{
    const int n = blockIdx.x / K;
    const int k = blockIdx.x % K;
    const int idx = threadIdx.x + blockIdx.y * blockDim.x;
    const int row = idx / (outW / 2) * 2;
    const int col = idx % (outW / 2) * 2;

    extern __shared__ float smem[];
    float* shared_f = smem;
    float m[16] = {0.0f};
    float im_tile[16], v_ncp[16];

    for(int c = threadIdx.x; ; c += blockDim.x){
        if(c >= C) break;
        const float* f = filter + (k * C + c) * 9;
        float* out = shared_f + c * 16;
        float g[9] = {f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8]};

        out[0] = g[0];
        out[1] = 0.5f * (g[0] + g[1] + g[2]);
        out[2] = 0.5f * (g[0] - g[1] + g[2]);
        out[3] = g[2];
        out[4] = 0.5f * (g[0] + g[3] + g[6]);
        out[5] = 0.25f * (g[0] + g[1] + g[2] + g[3] + g[4] + g[5] + g[6] + g[7] + g[8]);
        out[6] = 0.25f * (g[0] - g[1] + g[2] + g[3] - g[4] + g[5] + g[6] - g[7] + g[8]);
        out[7] = 0.5f * (g[2] + g[5] + g[8]);
        out[8] = 0.5f * (g[0] - g[3] + g[6]);
        out[9] = 0.25f * (g[0] + g[1] + g[2] - g[3] - g[4] - g[5] + g[6] + g[7] + g[8]);
        out[10] = 0.25f * (g[0] - g[1] + g[2] - g[3] + g[4] - g[5] + g[6] - g[7] + g[8]);
        out[11] = 0.5f * (g[2] - g[5] + g[8]);
        out[12] = g[6];
        out[13] = 0.5f * (g[6] + g[7] + g[8]);
        out[14] = 0.5f * (g[6] - g[7] + g[8]);
        out[15] = g[8];
    }
    __syncthreads();
    if(idx >= (outH / 2) * (outW / 2)) return;

    for(int c = 0; c < C; c++){
        for(int i = 0; i < 16; i++)
            im_tile[i] = image[((n * C + c) * H + (row + i / 4)) * W + (col + i % 4)];

        v_ncp[0] = im_tile[0] - im_tile[2] - im_tile[8] + im_tile[10];
        v_ncp[1] = im_tile[1] + im_tile[2] - im_tile[9] - im_tile[10];
        v_ncp[2] = -im_tile[1] + im_tile[2] + im_tile[9] - im_tile[10];
        v_ncp[3] = im_tile[1] - im_tile[3] - im_tile[9] + im_tile[11];
        v_ncp[4] = im_tile[4] - im_tile[6] + im_tile[8] - im_tile[10];
        v_ncp[5] = im_tile[5] + im_tile[6] + im_tile[9] + im_tile[10];
        v_ncp[6] = -im_tile[5] + im_tile[6] - im_tile[9] + im_tile[10];
        v_ncp[7] = im_tile[5] - im_tile[7] + im_tile[9] - im_tile[11];
        v_ncp[8] = -im_tile[4] + im_tile[6] + im_tile[8] - im_tile[10];
        v_ncp[9] = -im_tile[5] - im_tile[6] + im_tile[9] + im_tile[10];
        v_ncp[10] = im_tile[5] - im_tile[6] - im_tile[9] + im_tile[10];
        v_ncp[11] = -im_tile[5] + im_tile[7] + im_tile[9] - im_tile[11];
        v_ncp[12] = im_tile[4] - im_tile[6] - im_tile[12] + im_tile[14];
        v_ncp[13] = im_tile[5] + im_tile[6] - im_tile[13] - im_tile[14];
        v_ncp[14] = -im_tile[5] + im_tile[6] + im_tile[13] - im_tile[14];
        v_ncp[15] = im_tile[5] - im_tile[7] - im_tile[13] + im_tile[15];

        for(int a = 0; a < 16; a++)
            m[a] += v_ncp[a] * shared_f[c * 16 + a];
    }

    output[blockIdx.x * outH * outW + row * outW + col] = m[0] + m[1] + m[2] + m[4] + m[5] + m[6] + m[8] + m[9] + m[10];
    output[blockIdx.x * outH * outW + row * outW + (col + 1)] = m[1] - m[2] - m[3] + m[5] - m[6] - m[7] + m[9] - m[10] - m[11];
    output[blockIdx.x * outH * outW + (row + 1) * outW + col] = m[4] + m[5] + m[6] - m[8] - m[9] - m[10] - m[12] - m[13] - m[14];
    output[blockIdx.x * outH * outW + (row + 1) * outW + (col + 1)] = m[5] - m[6] - m[7] - m[9] + m[10] + m[11] - m[13] + m[14] + m[15];
}

void winograd_conv(thrust::device_vector<float>& image,
                   thrust::device_vector<float>& filter, 
                   thrust::device_vector<float>& out,
                   thrust::device_vector<float>& U,
                   thrust::device_vector<float>& V, 
                   thrust::device_vector<float>& M,
                   int H, int W, int C, int K, int N)
{
    const int outH = H - 2;
    const int outW = W - 2;

    int threads_per_block;
    int num_threads = (outH / 2) * (outW / 2);
    if(num_threads < 32){
        threads_per_block = 32;
    } else if(num_threads < 64){
        threads_per_block = 64;
    } else if(num_threads < 128){
        threads_per_block = 128;
    } else if(num_threads < 256){
        threads_per_block = 256;
    } else if(num_threads < 512){
        threads_per_block = 512;
    } else {
        threads_per_block = 1024;
    }
    dim3 grid_size = dim3(N * K, (num_threads + threads_per_block - 1) / threads_per_block);
    size_t smem_size = C * 4 * 4 * sizeof(float);
    winograd_conv_kernel<<<grid_size, threads_per_block, smem_size>>>(
        image.data().get(), filter.data().get(), out.data().get(),
        N, C, H, W, K, outH, outW
    );

    hipDeviceSynchronize();
}